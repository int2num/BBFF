#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void BFSor::copydata(int s,vector<edge>&edges,int nodenum){
};
void BFSor::dellocate(){
};
void BFSor::allocate(int maxn,int maxedge){
}
void BFSor::topsort()
{
};
void BFSor::updatE(vector<vector<int>>&esigns)
{
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<nein[i].size();j++)
			{
				/*if(k==21&&neie[i][j]==0)
					{
					cout<<"find it!!!!!!"<<endl;
					cout<<"value is :"<<esigns[k][neie[i][j]]<<endl;
					}*/
				if(esigns[k][neie[i][j]]<0)
					te[count]=-1;
				else
					te[count]=nein[i][j];
				/*if(k==21&&neie[i][j]==0)
				{
					cout<<"mao:"<<st[count]<<" "<<te[count]<<endl;
					cout<<"stid:"<<stid[count]<<endl;
				}*/
			    count++;
			}
	//cout<<"after:"<<st[count]<<" "<<te[count]<<endl;
	hipMemcpy(dev_te,te,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
};
void BFSor::updatS(vector<vector<Sot>>&stpair)
{
	L[0]=0;
	L[1]=LY1;
	L[2]=LY2;
	S[0]=stpair[0].size();
	S[1]=stpair[1].size();
	stps=stpair;
	int count=0;
	ncount=L[1]*S[0]+L[2]*S[1];
	memset(d,1,ncount*nodenum*sizeof(int));
	memset(p,-1,ncount*nodenum*sizeof(int));
	for(int k=0;k<L[1];k++)
		{
		for(int j=0;j<stpair[0].size();j++)
			{
			 d[count*nodenum+stpair[0][j].s]=0;
			 count++;
			}
		}
	for(int k=0;k<L[2];k++)
		{
		for(int j=0;j<stpair[1].size();j++)
			{
			 d[count*nodenum+stpair[1][j].s]=0;
			 count++;
			}
		}
	Size[0]=edges.size()*L[1]*S[0];
	Size[1]=edges.size()*L[2]*S[1];
	hipMemcpy(dev_d,d,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_p,p,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
}
void BFSor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,int _nodenum)
{
	//cout<<"in paraller BFS init"<<endl;
	nodenum=_nodenum;
	edges=ext.first;
	vector<vector<int>>esigns;
	esigns=ext.second;
	stp=stpair;
	mark=new int;
	*mark=0;
	W=WD+1;
	st=new int[edges.size()*LY];
	te=new int[edges.size()*LY];
	stid=new int[edges.size()*LY];
	d=new int[nodenum*LY*YE];
	p=new int[nodenum*LY*YE];
	esignes=new int[edges.size()*LY];
	vector<vector<int>>ein(nodenum*LY,vector<int>());
	neibn=ein;
	vector<vector<int>>eie(nodenum,vector<int>());
	neie=eie;
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			neibn[s].push_back(t);
			neie[s].push_back(i);
		}
	nein=neibn;
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]<0)
					te[count]=-1;
				else
					te[count]=neibn[i][j];
				stid[count]=neie[i][j];
				count++;
			}
	for(int i=0;i<nodenum*LY*YE;i++)
		d[i]=2*WD+1,p[i]=-1;
	hipMalloc((void**)&dev_st,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_stid,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_p,YE*LY*nodenum*sizeof(int));
	hipMemcpy(dev_te,te,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_stid,stid,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
};
BFSor::BFSor():L(PC+1,0),S(PC,0),NF(PC,0),Size(2,0)
{
};
__global__ void BFSfast(int *st,int *te,int *d,int* p,int *stid,int E,int N,int size,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid];
	if(t<0)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	if(d[s+off]==round-1&&d[t+off]>round)
		{	d[t+off]=round;
			//p[t+off]=stid[eeid];
		}
}
__global__ void BFScolor(int *st,int *te,int *d,int* p,int *stid,int E,int N,int size,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid];
	if(t<0)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	if(d[s+off]==d[t+off]-1)
		{	//d[t+off]=round;
			p[t+off]=stid[eeid];
		}
}
vector<vector<Rout>> BFSor::routalg(int s,int t,int bw)
{
	//cout<<"blasting "<<endl;
	int kk=1;
	time_t start,end;
	start=clock();
	int size=edges.size()*LY*YE;
	hipStream_t stream0;
	hipStreamCreate(&stream0);
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	int leoff=edges.size()*L[1];
	int nuoff=L[1]*S[0]*nodenum;
	for(int i=1;i<WD+1;i++)
		{
			BFSfast<<<Size[0]/512+1,512,0,stream0>>>(dev_st,dev_te,dev_d,dev_p,dev_stid,edges.size(),nodenum,Size[0],i,0,0,S[0],L[1]);
			BFSfast<<<Size[1]/512+1,512,0,stream1>>>(dev_st,dev_te,dev_d,dev_p,dev_stid,edges.size(),nodenum,Size[1],i,leoff,nuoff,S[1],L[2]);
		}
	BFScolor<<<Size[0]/512+1,512,0,stream0>>>(dev_st,dev_te,dev_d,dev_p,dev_stid,edges.size(),nodenum,Size[0],0,0,0,S[0],L[1]);
	BFScolor<<<Size[1]/512+1,512,0,stream1>>>(dev_st,dev_te,dev_d,dev_p,dev_stid,edges.size(),nodenum,Size[1],0,leoff,nuoff,S[1],L[2]);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream0);
	hipMemcpy(d,dev_d,ncount*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(p,dev_p,ncount*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	vector<vector<Rout>>result(2,vector<Rout>());
	int offer=L[1]*nodenum*stps[0].size();
	vector<int>LL(3,0);
	LL=L;
	LL[2]+=LL[1];
	int count=0;
	for(int y=1;y<PC+1;y++)
		for(int k=LL[y-1];k<LL[y];k++)
		{			
			for(int l=0;l<stps[y-1].size();l++)
			{	
				int s=stps[y-1][l].s;
				vector<int>ters=stps[y-1][l].ters;
				int off=count*nodenum;
				for(int i=0;i<ters.size();i++)
				{
					int id=stps[y-1][l].mmpid[ters[i]];
					int t=ters[i];
					int ds=d[off+t];
					if(ds>WD)continue;
					//cout<<k<<" "<<l<<" "<<s<<" "<<t<<" "<<ds<<" : "<<d[s+off]<<" "<<s+off<<endl;
					/*int prn=off+t;
					int hop=0;
					vector<int>rout;
					while(prn!=s+off)
					{
						int eid=p[prn];
						rout.push_back(eid);
						prn=edges[eid].s+off;
						//cout<<prn<<endl;
						hop++;
					}*/
					Rout S(s,t,id,ds,off,k);
					result[y-1].push_back(S);			
				}
				count++;
			}
		}
	end=clock();
	//cout<<"GPU time is : "<<end-start<<endl;
	//cout<<"over!"<<endl;
	//hipFree(dev_te);
	//hipFree(dev_st);
	//hipFree(dev_d);
	//cout<<"before return"<<endl;
	return result;
};
/*__global__ void BFSfast(int *st,int *te,int *d,int round,int E,int N,int size)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;
	int eid=(i%(E*LY));
	int s=st[eid],t=te[eid];
	int off=(i/(E*LY))*N+(eid/E)*N*YE;
	if(d[s+off]==round-1&&d[t+off]>round)
		d[t+off]=round;
}*/